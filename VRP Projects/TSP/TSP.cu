#include "hip/hip_runtime.h"
﻿// Special CUDA API headers
#include "hip/hip_runtime.h"
#include ""
#include ""
#include <hip/hip_cooperative_groups.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

// Custom header containing Control Panel
#include "TSP.cuh"

// General purpose headers
#include <iostream>
#include <stdbool.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>

// Cooperative groups namespace for block and grid sync
using namespace cooperative_groups;

// Main function
int main(int argc, char* argv[])
{
    // Variables used for reading from txt file
    FILE* pfile;    // File pointer
    int fileNameIdx;
    bool foundDistFile = false;   // Error handling
    bool foundRoute;
    int size;    // Number of graph vertices
    int i;  // Iterator
    srand(time(0)); // Need seeds for random solutions

    // Processing command line arguments
    for (i = 1; i < argc; ++i)
    {
        /// Distance file: REQUIRED
        // Command Line Syntax: ... --data [file_name]
        if ((strcmp(argv[i], "-d") == 0) || (strcmp(argv[i], "--data") == 0))
        {
            pfile = fopen(argv[++i], "r");
            if (pfile == NULL) {
                fprintf(stderr, "Unable to open file \"%s\"", argv[i]);
                return -1;
            }
            fileNameIdx = i;
            printf("Opening file \"%s\"!\n", argv[fileNameIdx]);
            foundDistFile = true;
        }

        /// Number of threads: OPTIONAL (default: 1024)
        // Command Line Syntax: ... --ants [number of ants]
        else if ((strcmp(argv[i], "-a") == 0) || (strcmp(argv[i], "--ants") == 0))
        {
            if (sscanf(argv[++i], "%d", &ants) != 1) {
                fprintf(stderr, "Unable to read ant number!\n");
            }
            else {
                printf("Given ant number : %d\n", ants);
            }
        }

        /// Number of full thread blocks: OPTIONAL
        // Command Line Syntax: ... --blocks [number of blocks]
        else if ((strcmp(argv[i], "-b") == 0) || (strcmp(argv[i], "--blocks") == 0))
        {
            if (sscanf(argv[++i], "%d", &ants) != 1) {
                fprintf(stderr, "Unable to read ant number!\n");
            }
            else {
                printf("Given block number : %d\n", ants);
                ants *= BLOCK_SIZE;
            }
        }
    }

    // Checking required elements
    if (!foundDistFile)
    {
        fprintf(stderr, "Please give a file in command line arguments to set the Distance Matrix!\n");
        fprintf(stderr, "Command Line Syntax:\n\t--data [data_file].txt\n");
        return -1;
    }

    // File syntax : 1st row must contain graph size in decimal
    // Following rows: graph edge values separated with comma (,)
    if (fscanf_s(pfile, "%d \n", &size) == 0) {
        fprintf(stderr, "Unable to read Size!\n Make sure you have the right file syntax!\n");
        fclose(pfile);
        return -1;
    }

    // Distance matrix
    // Store type: adjacency matrix format
    float* Dist = (float*)calloc(size * size, sizeof(float));

    // Reading distance values from dist file
    for (int ii = 0; ii < size; ++ii) {
        float temp;

        for (int jj = 0; jj < size; ++jj) {
            if (fscanf_s(pfile, "%f", &temp) == 0) {
                fprintf(stderr, "Error reading file \"%s\" distance(%d,%d)\n", argv[fileNameIdx], ii, jj);
                fclose(pfile);
                return -1;
            }
            Dist[ii * size + jj] = temp;
        }
        fscanf_s(pfile, "\n");
    }

    // Closing dist file
    printf("Closing file \"%s\"!\n", argv[fileNameIdx]);
    if (fclose(pfile) != 0) {
        fprintf(stderr, "Unable to close file \"%s\"!\n", argv[fileNameIdx]);
        return -1;
    }

    // Printing Matrix
    printf("Given Dist matrix:\n");
    print(Dist, size);

    // Host Variables

    TSP::CUDA_Main_ParamTypedef params;
    params.foundRoute = &foundRoute;
    params.antNum = ants;
    params.Dist = Dist;
    params.Pheromone = (float*)malloc(size * size * sizeof(float));
    params.route = (int*)malloc(size * sizeof(int));
    params.size = size;

    printf("Traveling Salesman Problem with Ant Colony Algorithm\n");
    TSP::CUDA_main(params);

    free(params.Dist);
    free(params.Pheromone);
    free(params.route);

    //getchar();
    return 0;
}

namespace TSP {

    // Global variables for multi grid Kernel
    __device__ Kernel_GlobalParamTypedef globalParams;

    // Host function for CUDA
    hipError_t CUDA_main(CUDA_Main_ParamTypedef h_params)
    {
        hipError_t cudaStatus;
        // Local variables
        int size = h_params.size;    // Number of graph vertices
        int antNum = h_params.antNum;    // Number of Ants (= threads) 

        if (!inputGood(&h_params)) {
            fprintf(stderr, "Invalid Input values!\n");
            return hipError_t::hipErrorInvalidConfiguration;
        }

        // Choosing GPU, may be nessesary in a multi-GPU system
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
            return cudaStatus;
        }

        // Calculates the number of Grid blocks to execute
        // Number of threads = number of ants
        int BlockNum = 1;
        if (antNum > BLOCK_SIZE) {
            BlockNum = my_ceil(antNum, BLOCK_SIZE);
            antNum = BlockNum * BLOCK_SIZE; // For better usage of parallel threads
        }

        // Device pointers
        Kernel_ParamTypedef d_kernelParams;
        d_kernelParams.Dist = NULL;
        d_kernelParams.foundRoute = NULL;
        d_kernelParams.Pheromone = NULL;
        d_kernelParams.route = NULL;
        d_kernelParams.state = NULL;
        d_kernelParams.antNum = antNum;
        d_kernelParams.size = size;
        d_kernelParams.state = NULL;

        // Config parameters
        Kernel_ConfigParamTypedef d_configParams;
        d_configParams.Rho = RHO;
        d_configParams.Follower_Generations = FOLLOWER_GENERATIONS;
        d_configParams.Initial_Pheromone_Value = INITIAL_PHEROMONE_VALUE;
        d_configParams.maxTryNumber = size;
        d_configParams.Random_Generations = RANDOM_GENERATIONS;
        d_configParams.Repetitions = REPETITIONS;
        d_configParams.Reward_Multiplier = REWARD_MULTIPLIER;

        // Size of device malloc
        size_t Dist_bytes = size * size * sizeof(float);
        size_t route_bytes = size * sizeof(int);
        size_t foundRoute_bytes = sizeof(bool); // May be optimized, only for better transparency
        size_t antRoute_bytes = antNum * size * sizeof(int);
        size_t state_bytes = antNum * sizeof(hiprandState);
        // CUDA Malloc

        // Dist
        cudaStatus = hipMalloc((void**)&d_kernelParams.Dist, Dist_bytes);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "d_Dist hipMalloc failed!\n");
            Free_device_memory(d_kernelParams);
            return cudaStatus;
        }
        // Pheromone
        cudaStatus = hipMalloc((void**)&d_kernelParams.Pheromone, Dist_bytes);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "d_Pheromone hipMalloc failed!\n");
            Free_device_memory(d_kernelParams);
            return cudaStatus;
        }
        // route
        cudaStatus = hipMalloc((void**)&d_kernelParams.route, route_bytes);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "d_Route hipMalloc failed!\n");
            Free_device_memory(d_kernelParams);
            return cudaStatus;
        }
        // foundRoute : flag
        cudaStatus = hipMalloc((void**)&d_kernelParams.foundRoute, foundRoute_bytes);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "d_Route hipMalloc failed!\n");
            Free_device_memory(d_kernelParams);
            return cudaStatus;
        }
        // antRoute : auxiliary array
        cudaStatus = hipMalloc((void**)&d_kernelParams.antRoute, antRoute_bytes);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "antRoute hipMalloc failed!\n");
            Free_device_memory(d_kernelParams);
            return cudaStatus;
        }
        // state : CUDA supported random seeds for threads
        cudaStatus = hipMalloc(&d_kernelParams.state, state_bytes);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!\n");
            Free_device_memory(d_kernelParams);
            return cudaStatus;
        }

        // Copying data : Host -> Device
        cudaStatus = hipMemcpy(d_kernelParams.Dist, h_params.Dist, Dist_bytes, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Dist hipMemcpy failed!\n");
            Free_device_memory(d_kernelParams);
            return cudaStatus;
        }

        printf("Called function with %d Block", BlockNum);
        if (BlockNum == 1)
            printf(": \n");
        else
            printf("s: \n");
        int threadPerBlock = (antNum > BLOCK_SIZE) ? BLOCK_SIZE : antNum;

        // setup seeds

        setup_kernel << < BlockNum, threadPerBlock >> > (d_kernelParams.state, time(NULL) * rand());

        // Kernel call

        float min = FLT_MAX;
        float sum = 0.0f;
        int foundCount = 0;

        for (int iter = 0; iter < SERIALMAXTRIES; iter++)
        {
            printf("\nAttempt #%d ||\n", iter);

            if (BlockNum == 1) {
                Kernel_1Block <<< 1, threadPerBlock >>> (d_kernelParams, d_configParams);
            }
            else
            {
                // During Kernel call it's important to use hipLaunchCooperativeKernel CUDA runtime launch API
                // or its CUDA driver equivalent instead of the <<<...>>> syntax

                // Sets supportsCoopLaunch=1 if the operation is supported on device 0
                // Only compute capability 6.0 or higher!
                int dev = 0;
                int supportsCoopLaunch = 0;
                hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);
                if (supportsCoopLaunch != 1)
                {
                    fprintf(stderr, "Cooperative Launch is not supported on this machine configuration.");
                    Free_device_memory(d_kernelParams);
                    return cudaStatus;
                }

                // Call arguments
                void* kernelArgs[] = { &d_kernelParams, &d_configParams };

                hipLaunchCooperativeKernel((void*)Kernel_multiBlock, BlockNum, BLOCK_SIZE, kernelArgs);
            }

            // Error handling during Kernel execution
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "AntKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
                Free_device_memory(d_kernelParams);
                return cudaStatus;
            }

            // hipDeviceSynchronize waits for the kernel to finish
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching antKernel!\n", cudaStatus);
                Free_device_memory(d_kernelParams);
                return cudaStatus;
            }

            // Copying processed data from GPU device
            cudaStatus = hipMemcpy(h_params.route, d_kernelParams.route, route_bytes, hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "route dev->host hipMemcpy failed!");
                // Frees GPU device memory
                Free_device_memory(d_kernelParams);
                return cudaStatus;
            }
            cudaStatus = hipMemcpy(h_params.foundRoute, d_kernelParams.foundRoute, sizeof(bool), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "foundRoute flag dev->host hipMemcpy failed!");
                // Frees GPU device memory
                Free_device_memory(d_kernelParams);
                return cudaStatus;
            }

            float _length = sequencePrint(h_params.route, h_params.Dist, size);
            if (_length > 0) {
                foundCount++;
                sum += _length;
                if (_length < min)
                    min = _length;
            }
        }
        printf("\nSummary:\nAverage length: %.2f\n", sum / foundCount);
        printf("Minimal length: %.2f\n", min);

        // Frees GPU device memory
        Free_device_memory(d_kernelParams);
        return cudaStatus;

    }

    // Testing input for main CUDA function
    // Returns true if input data syntax is good
    // Disclaimer: Only tests NULL property of pointers, does not 100% guarantee perfect data
    __host__ __device__ inline bool inputGood(CUDA_Main_ParamTypedef* params) {
        return (
            32 <= params->antNum &&    // At least 32 threads (for GPU usage)
            2 <= params->size &&      // At least 2 nodes
            NULL != params->Dist &&
            NULL != params->foundRoute &&
            NULL != params->Pheromone &&
            NULL != params->route);
    }

    // Testing input for main CUDA function
    // Returns true if input data syntax is good
    // Disclaimer: Only tests NULL property of pointers, does not 100% guarantee perfect data
    __device__ inline bool inputGood(Kernel_ParamTypedef* params) {
        return (
            32 <= params->antNum &&    // At least 32 threads (for GPU usage)
            2 <= params->size &&      // At least 2 nodes
            NULL != params->Dist &&
            NULL != params->foundRoute &&
            NULL != params->Pheromone &&
            NULL != params->route);
    }

    // Frees device memory with hipFree if pointer is not NULL
    // Important: all initial values of device pointers MUST BE NULL
    void Free_device_memory(Kernel_ParamTypedef params) {
        // Temporary device data structures
        if (NULL != params.Dist) hipFree(params.Dist);
        if (NULL != params.antRoute) hipFree(params.antRoute);
        if (NULL != params.Pheromone) hipFree(params.Pheromone);
        if (NULL != params.route) hipFree(params.route);
        if (NULL != params.state) hipFree(params.state);
    }

    __device__ __host__ float sequencePrint(int* route, float* Dist, int size) {
        if (NULL == route || NULL == Dist || 2 > size) {
            printf("Invalid input of sequencePrint!\n");
            return -1;
        }

        // Check for dead end
        for (int i = 0; i < size; ++i)
        {
            int src = route[i];
            int dst = route[(i + 1) % size];
            assert(src > -1 && src < size&& dst > -1 && dst < size);
            if (Dist[src * size + dst] < 0)
            {
                printf("Route not possible!\n");
                return -1;
            }
        }

        printf("Sequence : ");
        float l = 0;
        for (int i = 0; i < size; ++i)
        {
            int src = route[i];
            int dst = route[(i + 1) % size];
            printf("%d ", src);
            l += Dist[src * size + dst];
        }
        printf("%d\n", route[0]);
        printf(" Total length : %.2f\n ", l);
        return l;
    }

    // Initializes a random seed for each different threads
    __global__ void setup_kernel(hiprandState* state, unsigned long seed)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, id, id, &state[id]);
    }


    // 1 block sized kernel
    __global__ void Kernel_1Block(
        Kernel_ParamTypedef params,
        Kernel_ConfigParamTypedef configParams
    )
    {
        // Dist (i,j) means the distance from vertex i to vertex j
        // If no edge drawn between them: Dist(i,j) = -1 (expected syntax)
        thread_block block = this_thread_block();

        int antIndex = threadIdx.x;  // Ant index 0 - (antNum-1)

        if (antIndex >= params.antNum || blockIdx.x > 0)     // Defense against overaddressing
            return;

        // Shared variables between threads in the same block
        __shared__ bool invalidInput;       // Variable used to detecting invalid input
        __shared__ bool isolatedVertex;     // Variable used to detecting isolated vertex (for optimization purposes)
        __shared__ float averageDist;    // Average edge distance
        __shared__ float multiplicationConst;
        __shared__ int size;                // Local Copy of argument parameter

        // Initialization of temporary variables
        invalidInput = false;
        isolatedVertex = false;
        averageDist = 0.0f;
        multiplicationConst = 0.0f;
        size = params.size; // Needs to be written too many times
        *params.foundRoute = false;
        globalParams.minRes = FLT_MAX;

        // Invalidate route vector
        for (int i = 0; i < size; i++)
            params.route[i] = 0;

        // Input check
        if (antIndex == 0 && !inputGood(&params)) {
            invalidInput = true;
            printf("Invalid Input\n");
        }
        block.sync();

        // Pheromone matrix initialization
        if (antIndex == 0)
        {
            bool foundNeighboor = false;    // Checking if any of the nodes are isolated
            int i, j;
            for (i = 0; i < size; i++) {
                for (j = 0; j < size; j++) {
                    // Initializing Pheromone graph (anti - unitmatrix, all main diagonal elements are 0)
                    // 0 Pheromone value if no edge drawn
                    // Initial Pheromone value is of consideration in the Control panel
                    if ((i == j) || (params.Dist[i * size + j] < 0))
                        params.Pheromone[i * size + j] = 0.0f;
                    else
                        params.Pheromone[i * size + j] = configParams.Initial_Pheromone_Value;

                    // Error handling 
                    // Check if there are invalid given elements 
                    // Valid input if: positive OR -1 OR 0 (only if i=j)
                    if (i != j && params.Dist[i * size + j] <= 0
                        && params.Dist[i * size + j] != -1)
                    {
                        printf("Dist(%d,%d) incorrect!\n", i, j);
                        invalidInput = true;
                        break;
                    }
                    if (!foundNeighboor && params.Dist[i * size + j] > 0) {
                        // Has neighboor therefore not isolated
                        foundNeighboor = true;
                    }
                }
                if (!foundNeighboor) { // Did not have any neighboors => wrong model of TSP
                    printf("Vertex %d isolated!\n", i);
                    isolatedVertex = true;
                }
            }
        }

        block.sync();

        if (invalidInput || isolatedVertex) {   // Invalid input, so no point of continuing
            return;                             // Case of isolated node means no route exists
        }

        // Case of only 2 nodes: handle quickly in 1 thread
        if (size == 2) {
            if (antIndex == 0) {
                if (params.Dist[0 * size + 1] > 0 && params.Dist[1 * size + 0] > 0) {    // Route exists
                    *params.foundRoute = true;
                    params.route[0] = 0;    // Route = [0 1]
                    params.route[1] = 1;
                }
            }
            block.sync();
            return;
        }

        // Left: Connected(?) graph with at least 3 vertices

        // Calculating average distance
        if (antIndex == 0) {
            float sum = 0.0f;   // Sum of edge values
            int numPos = 0;     // Number of edges
            for (int i = 0; i < size; i++) {
                for (int j = 0; j < size; j++)
                {
                    float edge = params.Dist[i * size + j];
                    if (edge > 0)
                    {
                        sum += edge;
                        numPos++;
                    }
                }
            }
            averageDist = sum / numPos * size;

        }
        block.sync();

        // Default values for routes
        initAntRoute(&params, antIndex);
        block.sync();

        // Ants travelling to all directions
        for (int repNumber = 0; repNumber < configParams.Repetitions; repNumber++)
        {
            if (antIndex == 0)
                multiplicationConst = averageDist / configParams.Rho * 5;
            block.sync();

            // Trying for every possible second vertices
            for (int secondVertex = 1; secondVertex < size; secondVertex++)
            {
                generateRandomSolution(&params, antIndex, secondVertex);
                // Evaluating the given solution: modifies Pheromone matrix more if shorter path found
                evaluateSolution(&params, antIndex, multiplicationConst, configParams.Reward_Multiplier, repNumber);
                block.sync();
            }

            // Numerous random guesses
            for (int j = 0; j < configParams.Random_Generations; j++) {
                // Random second vertices
                generateRandomSolution(&params, antIndex, -1);
                evaluateSolution(&params, antIndex, multiplicationConst, configParams.Reward_Multiplier, repNumber);
                block.sync();
            }

            if (antIndex == 0)
                multiplicationConst *= 2;
            block.sync();

            // Lots of ants following pheromone of previous ants
            for (int gen = 0; gen < configParams.Follower_Generations; gen++) {

                // Reducing previous pheromon values by value RHO (modifiable in the Control Panel)
                if (antIndex == 0) {
                    for (int i = 0; i < size; i++) {
                        for (int j = 0; j < size; j++)
                            params.Pheromone[i * size + j] *= configParams.Rho;
                    }
                }
                block.sync();

                // New ants following pheromone of previous ants
                followPheromones(&params, antIndex, configParams.maxTryNumber);
                block.sync();
                evaluateSolution(&params, antIndex, multiplicationConst, configParams.Reward_Multiplier, repNumber);
                block.sync();
            }
        }

        // Removing unwanted threads
        if (antIndex != 0)
            return;

        // Choosing path with greedy algorithm if we dont have a valid answer
        if (!validRoute(&params)) {
            // Mostly occurs when it did not find any routes, but we also prepare for corrupted memory
            printf("Need to find route in greedy mode!\n");
            greedySequence(&params);
        }
        else {
            // We already found a solution but check the pheromones just in case
            greedySequence(&params, 0);
            evaluateSolution(&params, 0, 1.0f, 1.0f, 1);
        }
        // We found a route if given length is greater than zero
        *params.foundRoute = (antRouteLength(&params, 0) > 0);
    }

    // Multiblock sized kernel
    __global__ void Kernel_multiBlock(
        Kernel_ParamTypedef params,
        Kernel_ConfigParamTypedef configParams)
    {
        // Dist (i,j) means the distance from vertex i to vertex j
        // If no edge drawn between them: Dist(i,j) = -1 (expected syntax)
        grid_group grid = this_grid();
        if (!grid.is_valid())
            return;
        grid.sync();
        int antIndex = blockIdx.x * blockDim.x + threadIdx.x;  // ant index
        grid.sync();

        float multiplicationConst;

        // Initialization
        globalParams.invalidInput = false;
        globalParams.isolatedVertex = false;
        globalParams.averageDist = 0.0f;
        multiplicationConst = 0.0f;
        *params.foundRoute = false;
        globalParams.minRes = FLT_MAX;

        // Invalidate route vector
        for (int i = 0; i < params.size; i++)
            params.route[i] = 0;

        // Input check
        if (antIndex == 0 && !inputGood(&params)) {
            globalParams.invalidInput = true;
            printf("Invalid Input\n");
        }
        grid.sync();

        // Pheromone matrix initialization
        if (antIndex == 0)
        {
            bool foundNeighboor = false;    // Checking if any of the nodes are isolated
            int i, j;
            for (i = 0; i < params.size; i++) {
                for (j = 0; j < params.size; j++) {
                    // Initializing Pheromone graph (anti - unitmatrix, all main diagonal elements are 0)
                    // 0 Pheromone value if no edge drawn
                    // Initial Pheromone value is of consideration in the Control panel
                    if ((i == j) || (params.Dist[i * params.size + j] < 0))
                        params.Pheromone[i * params.size + j] = 0;
                    else
                        params.Pheromone[i * params.size + j] = configParams.Initial_Pheromone_Value;

                    // Error handling 
                    // Check if there are invalid given elements 
                    // Valid input if: positive OR -1 OR 0 (only if i=j)
                    if (i != j && params.Dist[i * params.size + j] <= 0
                        && params.Dist[i * params.size + j] != -1)
                    {
                        printf("Dist(%d,%d) incorrect!\n", i, j);
                        globalParams.invalidInput = true;
                        break;
                    }
                    if (!foundNeighboor && params.Dist[i * params.size + j] > 0) {
                        // Has neighboor therefore not isolated
                        foundNeighboor = true;
                    }
                }
                if (!foundNeighboor) { // Did not have any neighboors => wrong model of TSP
                    printf("Vertex %d isolated!\n", i);
                    globalParams.isolatedVertex = true;
                }
            }
        }
        grid.sync();

        if (globalParams.invalidInput || globalParams.isolatedVertex) {   // Invalid input, so no point of continuing
            return;                             // Case of isolated node means no route exists
        }

        // Case of only 2 nodes: handle quickly in 1 thread
        if (params.size == 2) {
            if (antIndex == 0) {
                if (params.Dist[0 * params.size + 1] > 0 && params.Dist[1 * params.size + 0] > 0)
                {    // Route exists
                    *params.foundRoute = true;
                    params.route[0] = 0;    // Route = [0 1]
                    params.route[1] = 1;
                }
            }
            grid.sync();
            return;
        }

        // Left: Connected(?) graph with at least 3 nodes
        // Calculating average distance
        __shared__ float sum;   // Sum of edge values
        sum = 0.0f;
        __shared__ int numPos;  // Number of edges
        numPos = 0;
        __shared__ float edge;  // Temp variable
        grid.sync();
        if (antIndex == 0)
        {
            for (int i = 0; i < params.size; i++) {
                for (int j = 0; j < params.size; j++)
                {
                    edge = params.Dist[i * params.size + j];
                    if (edge > 0)
                    {
                        sum += edge;
                        numPos++;
                    }
                }
            }
            globalParams.averageDist = sum / numPos * params.size;
        }
        grid.sync();

        // Initializing ant Routes 
        initAntRoute(&params, antIndex);
        grid.sync();

        // Ants travelling to all directions
        for (int repNumber = 0; repNumber < configParams.Repetitions; repNumber++)
        {
            multiplicationConst = globalParams.averageDist / configParams.Rho * 5.0f;

            // Trying for every possible second vertices
            for (int secondVertex = 1; secondVertex < params.size; secondVertex++)
            {
                generateRandomSolution(&params, antIndex, secondVertex);
                // Evaluating the given solution: modifies Pheromone matrix more if shorter path found
                evaluateSolution(&params, antIndex, multiplicationConst, configParams.Reward_Multiplier, repNumber);
                grid.sync();
            }

            // Numerous random guess
            for (int j = 0; j < configParams.Random_Generations; j++)
            {
                // Seconvertex = -1 means no prescribed second vertex
                generateRandomSolution(&params, antIndex, -1);
                grid.sync();

                // Evaluating the given solution: modifies Pheromone matrix more if shorter path found
                evaluateSolution(&params, antIndex, multiplicationConst, configParams.Reward_Multiplier, repNumber);
                grid.sync();
            }

            multiplicationConst *= 2;
            grid.sync();

            // Lots of ants following pheromone of previous ants
            for (int gen = 0; gen < configParams.Follower_Generations; gen++)
            {
                // Reducing previous pheromon values by value RHO (modifiable in the Control Panel)
                if (antIndex == 0) {
                    for (int i = 0; i < params.size; i++) {
                        for (int j = 0; j < params.size; j++)
                            params.Pheromone[i * params.size + j] *= configParams.Rho;
                    }
                }
                grid.sync();

                // New ants following pheromone of previous ants
                followPheromones(&params, antIndex, configParams.maxTryNumber);
                grid.sync();
                evaluateSolution(&params, antIndex, multiplicationConst, configParams.Reward_Multiplier, repNumber);
                grid.sync();
            }
        }

        if (antIndex == 0) {
            // Choosing path with greedy algorithm if we dont have a valid answer
            if (!validRoute(&params)) {
                printf("Need to find route in greedy mode!\n");
                greedySequence(&params);
            }
            else {
                // We already found a solution but check the pheromones just in case
                greedySequence(&params, 0);
                evaluateSolution(&params, 0, 1.0f, 1.0f, 1);
            }
        }


        grid.sync();   // We found a route if given length is greater than zero

        *params.foundRoute = (antRouteLength(&params, 0) > 0);
    }

    // Gets initial value of Route arrays
    __device__ void initAntRoute(
        Kernel_ParamTypedef* pkernelParams,
        int antIndex
    )
    {
        // Route init [0, 1, 2 ... size-1]
        int* antRouteOffset = pkernelParams->antRoute + antIndex * pkernelParams->size;   // Optimizing array addressing
        for (int idx = 0; idx < pkernelParams->size; idx++) {
            antRouteOffset[idx] = idx;
        }
    }


    inline __device__ bool isValidSecondVertex(int secondVertex, int size) {
        return (secondVertex > 0 && secondVertex < size);
    }

    // Generates a random sequence of numbers between 0 and (size - 1) starting with 0
    // secondVertex: Variable used for giving an arbitrary second vertex
    //      0 < secondvertex < size : valid input (condition = 1)
    //      else: invalid input, no mandatory second vertex (condition = 0)
    __device__ void generateRandomSolution(
        Kernel_ParamTypedef* pkernelParams,
        int antIndex,
        int secondVertex
    )
    {
        int* antRouteOffset = pkernelParams->antRoute + antIndex * pkernelParams->size;   // Optimizing array addressing
        // Expected to start in node 0 (in normal use this is already set, but for safety it's here)
        antRouteOffset[0] = 0;

        int min_rand_int = 1, max_rand_int = pkernelParams->size - 1;
        if (isValidSecondVertex(secondVertex, pkernelParams->size)) {
            min_rand_int = 2;
            int secVertexidx;
            // Find secondvertex in route
            for (secVertexidx = 0; secVertexidx < pkernelParams->size && antRouteOffset[secVertexidx] != secondVertex; ++secVertexidx);
            if (secVertexidx == pkernelParams->size)   // Could not find it, something went wrong, so we must order back the sequence
            {
                // If everything is correct, we may never enter here,
                // but in case so, we reconfigure the antRoute to default
                //printf("Error occured while generating random sequence: second vertex (%d) lost!\n", secondVertex);
                for (int idx = 2; idx < pkernelParams->size; idx++)
                    antRouteOffset[idx] = idx;

                antRouteOffset[1] = secondVertex;
                antRouteOffset[secondVertex] = 1;
            }
            else   // Second vertex found
            {
                antRouteOffset[secVertexidx] = antRouteOffset[1];
                antRouteOffset[1] = secondVertex;
            }
        }

        // n db random swap in the sequence, to shuffle the edges
        // executing [size] times random swaps
        // min_rand_int means the lower limit for the swap range
        // -> if there is an exact 2.vertex, then only the (3. - size.) vertex sequence needs to be changed
        for (int idx = min_rand_int; idx < pkernelParams->size; idx++)
        {
            float myrandf;
            int myrand;

            myrandf = hiprand_uniform(&pkernelParams->state[antIndex]);  // RND Number between 0 and 1
            myrandf *= (max_rand_int - min_rand_int + 0.999999f);
            myrandf += min_rand_int;
            myrand = (int)truncf(myrandf);

            assert(myrand <= max_rand_int);
            assert(myrand >= min_rand_int);

            int temp = antRouteOffset[idx];
            antRouteOffset[idx] = antRouteOffset[myrand];
            antRouteOffset[myrand] = temp;
        }
    }

    // Returns bool value of whether newParam is already listed in the route
    // Special care for node 0, which can be in the route [maxVehicles] times.
    // antindex = -1 means we are meant to look for the route vector
    __device__ bool alreadyListed(
        Kernel_ParamTypedef* pkernelParams,
        int antIndex,
        int idx,    // serial number of node in route
        int newParam
    )
    {
        assert(idx < pkernelParams->size);
        if (idx >= pkernelParams->size)
            return true;    // Rather make infinite cycle than overaddressing

        // Special care for -1: watching route vector
        if (antIndex == -1)
        {
            for (int i = 0; i < idx; ++i)
                if (newParam == pkernelParams->route[i])
                    return true;
            return false;
        }


        for (int i = 0; i < idx; ++i)
            if (newParam == pkernelParams->antRoute[antIndex * pkernelParams->size + i])
                return true;
        return false;
    }

    // Returns the length of the given route
    // Returns -1 if route not possible (for example has dead end)
    __device__ float antRouteLength(Kernel_ParamTypedef* pkernelParams, int antIndex)
    {
        int* antRouteOffset = pkernelParams->antRoute + antIndex * pkernelParams->size;   // Optimizing array addressing
        float length = 0;  // Return value
        int src, dst;

        for (int i = 0; i < pkernelParams->size; ++i)
        {
            src = antRouteOffset[i];
            dst = antRouteOffset[(i + 1) % pkernelParams->size];   // Next node

            float edgeLength = pkernelParams->Dist[src * pkernelParams->size + dst];
            if (edgeLength < 0) {
                return -1;
            }
            else {
                length += edgeLength;
            }
        }
        assert(length != 0);
        if (length == 0)
            return -1;

        return length;
    }


    // Represents az ant who follows other ants' pheromones
    // Generates a route with Roulette wheel method given the values of the Pheromone matrix
    __device__ void followPheromones(
        Kernel_ParamTypedef* pkernelParams,
        int antIndex,
        int maxTryNumber
    )
    {
        int* antRouteOffset = pkernelParams->antRoute + antIndex * pkernelParams->size;   // Optimizing array addressing
        hiprandState* statePtr = &(pkernelParams->state[antIndex]);
        // Expected to start in vertex 0
        pkernelParams->antRoute[antIndex * pkernelParams->size + 0] = 0;

        float sumPheromone = 0.0f;  // Weighted Roulette wheel: first we calculate the sum of weights
        for (int i = 0; i < pkernelParams->size; i++)
            sumPheromone += pkernelParams->Pheromone[i];

        // Starting from 2nd element of the Route
        for (int i = 1; i < pkernelParams->size; ++i)
        {
            int source = antRouteOffset[i - 1];   // Previous node
            int newParam;   // Variable for new route element
            bool foundVertexByRoulette = false;
            for (int j = 0; j < maxTryNumber && !foundVertexByRoulette; j++)
            {
                // RND Number between 0 and sumPheromone
                float myrandflt = hiprand_uniform(statePtr) * sumPheromone;
                float temp = pkernelParams->Pheromone[source * pkernelParams->size + 0]; // Used to store the matrix values

                for (newParam = 0; newParam < pkernelParams->size - 1; newParam++)
                {
                    if (myrandflt < temp)   // If newparam == size-1 then no other node to choose
                        break;
                    temp += pkernelParams->Pheromone[source * pkernelParams->size + newParam + 1];
                }   // If not already listed then adding to the sequence
                foundVertexByRoulette = !alreadyListed(pkernelParams, antIndex, i, newParam);
            }
            if (!foundVertexByRoulette)
            {
                // Next vertex choosen by equal chances
                do {
                    float newfloat = hiprand_uniform(statePtr);      // RND Number between 0 and 1
                    newfloat *= (pkernelParams->size - 1) + 0.999999f;  // Transforming into the needed range
                    newParam = (int)truncf(newfloat);
                } while (alreadyListed(pkernelParams, antIndex, i, newParam));
            }
            // At last the new vertex
            antRouteOffset[i] = newParam;
        }
    }

    // Manipulating the pheromone values according to the given solution
    // The longer the route is, the smaller amount we are adding
    // Sets the route vector if we found a best yet solution
    __device__ void evaluateSolution(
        Kernel_ParamTypedef* pkernelParams,
        int antIndex,
        float multiplConstant,
        float rewardMultiplier,
        int repNumber
    )
    {
        float length = antRouteLength(pkernelParams, antIndex);
        assert(length != 0);
        float additive = multiplConstant / length; // The longer the route is, the smaller amount we are adding
        if (length < globalParams.minRes && length > 0) { // Rewarding the ant with the best yet route
            // printf("New min found: %f, rep: %d\n", length, repNumber);   // If you are interested you can comment out
            copyAntRoute(pkernelParams, antIndex);
            globalParams.minRes = length;
            if (repNumber > 2)
                additive *= rewardMultiplier * (repNumber + 1) * (repNumber + 1);
        }

        // Route valid if length > 0
        if (length > 0) {
            int* antRouteOffset = pkernelParams->antRoute + antIndex * pkernelParams->size;   // Optimizing array addressing
            for (int i = 0; i < pkernelParams->size; i++)
            {
                int src = antRouteOffset[i];
                int dst = antRouteOffset[(i + 1) % pkernelParams->size];
                float* ptr = &(pkernelParams->Pheromone[src * pkernelParams->size + dst]);

                atomicAdd(ptr, additive);
            }
        }
    }

    // Auxilary function for greedy sequence
    // Return the highest vertex index not yet chosen
    /// row : row of previous route element (decides, which row to watch in the function)
    __device__ int maxInIdxRow(Kernel_ParamTypedef* pkernelParams, int row, int idx, int antIndex) {
        int maxidx = -1;
        float max = 0.0f;
        for (int i = 0; i < pkernelParams->size; i++)
        {
            // Go through the row elements to find the highest
            float observed = pkernelParams->Pheromone[row * pkernelParams->size + i];

            if (observed > max && !alreadyListed(pkernelParams, antIndex, idx, i))
            {
                max = observed;
                maxidx = i;
            }
        }
        //printf("%d. vertex with value of %.2f : %d\n", idx, max, maxidx);

        return maxidx;
    }

    // Generates a sequnce using greedy algorithm
    // Always chooses the highest possible value for the next vertex
    __device__ void greedySequence(Kernel_ParamTypedef* pkernelParams, int antIndex)
    {
        int* antRouteOffset = pkernelParams->antRoute
            + antIndex * pkernelParams->size;   // Optimizing array addressing
        // Special care for -1: watching route vector
        if (antIndex == -1)
            antRouteOffset = pkernelParams->route;

        antRouteOffset[0] = 0;
        for (int i = 1; i < pkernelParams->size; i++)
        {
            int node = antRouteOffset[i] = maxInIdxRow(pkernelParams, antRouteOffset[i - 1], i, antIndex);
            assert(node != -1);
        }
    }

    // Copies a route into the answer vector
    __device__ void copyAntRoute(Kernel_ParamTypedef* pkernelParams, int antIndex) {
        // Optimizing array addressing
        int* antRouteOffset = pkernelParams->antRoute + antIndex * pkernelParams->size;
        for (int i = 1; i < pkernelParams->size; i++)
            pkernelParams->route[i] = antRouteOffset[i];
    }

    // Finds a value in the route vector
    __device__ bool routeContain(Kernel_ParamTypedef* pkernelParams, int value)
    {
        for (int i = 1; i < pkernelParams->size; i++)
            if (pkernelParams->route[i] == value)
                return true;
        return false;
    }

    // Validates the output vector
    __device__ bool validRoute(Kernel_ParamTypedef* pkernelParams) {
        if (pkernelParams->route[0] != 0)
        {
            return false;
        }

        for (int i = 1; i < pkernelParams->size; i++)
        {
            if (!routeContain(pkernelParams, i))
            {
                return false;
            }
        }

        return true;
    }
}